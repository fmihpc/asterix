#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "../include/genericTsPool.h"
#include "../include/tinyAI.h"
#include <algorithm>
#include <chrono>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <fstream>
#include <iomanip>
#include <nvToolsExt.h>
#include <random>
#include <vector>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

using namespace TINYAI;
using namespace GENERIC_TS_POOL;
using namespace NumericMatrix;
using type_t = float;
using pixel = uint8_t;


struct Image {
   int width;
   int height;
   int channels;
   void* data;
};

template <typename T>
T rand_normal() {
   return (T)rand() / RAND_MAX * T(2.0) - T(1.0);
}

template <typename T>
NumericMatrix::HostMatrix<T> generate_fourier_features(NumericMatrix::HostMatrix<T>& input,

                                                       std::size_t num_features, T scale) {
   if (num_features == 0) {
      return NumericMatrix::HostMatrix<T>(input);
   }
   assert(num_features % 2 == 0);
   const std::size_t input_dims = input.ncols();
   // Construct B
   NumericMatrix::HostMatrix<T> B(input_dims, num_features);

   std::mt19937 rng(std::chrono::steady_clock::now().time_since_epoch().count());
   std::uniform_real_distribution<T> dist(-1.0, 1.0);
   for (std::size_t i = 0; i < input_dims; ++i) {
      for (std::size_t j = 0; j < num_features; ++j) {
         B(i, j) = scale * dist(rng); // rand_normal<T>();
      }
   }

   // Apply mapping
   NumericMatrix::HostMatrix<T> output(input.nrows(), 2 * num_features);
   for (std::size_t i = 0; i < input.nrows(); ++i) {
      for (std::size_t j = 0; j < num_features; ++j) {
         T dot_product = 0.0;
         for (std::size_t k = 0; k < input.ncols(); ++k) {
            dot_product += input(i, k) * B(k, j);
         }
         output(i, j) = std::sin(2.0 * M_PI * dot_product);
         output(i, j + num_features) = std::cos(2.0 * M_PI * dot_product);
      }
   }
   return output;
}

double calculate_mse(const unsigned char* original, const unsigned char* reconstructed, int width, int height) {
   double mse = 0.0;
   int num_pixels = width * height;
   for (int i = 0; i < num_pixels; i++) {
      int diff = original[i] - reconstructed[i];
      mse += diff * diff;
   }
   mse /= num_pixels;
   return mse;
}

double calculate_psnr(const unsigned char* original, const unsigned char* reconstructed, int width, int height) {
   double mse = calculate_mse(original, reconstructed, width, height);
   if (mse == 0) {
      return INFINITY;
   }
   double max_pixel_value = 255.0;
   double psnr = 20.0 * std::log10(max_pixel_value / std::sqrt(mse));
   return psnr;
}

int main(int argc, char** argv) {

   if (argc != 2) {
      fprintf(stderr, "ERROR: usage ./%s <image file>\n", argv[0]);
      return 1;
   }

   size_t N = 1ul * 1024ul * 1024ul * 1024ul;
#ifdef USE_GPU
   constexpr auto HW = BACKEND::DEVICE;
   void* mem;
   hipMalloc(&mem, N);
#else
   constexpr auto HW = BACKEND::HOST;
   void* mem = (void*)malloc(N);
#endif
   assert(mem && "Could not allocate memory !");
   GENERIC_TS_POOL::MemPool p(mem, N);

   size_t BATCHSIZE = 128;
   {

      const char* image_name = argv[1];
      Image img, rec_img;
      img.data = stbi_load(image_name, &img.width, &img.height, &img.channels, 0);
      rec_img.data = stbi_load(image_name, &rec_img.width, &rec_img.height, &rec_img.channels, 0);
      const size_t n_samples = img.width * img.height;
      NumericMatrix::HostMatrix<type_t> pos_temp(n_samples, 2);
      NumericMatrix::Matrix<type_t, HW> val(n_samples, 1, &p);
      NumericMatrix::Matrix<type_t, HW> recon(n_samples, 1, &p);
      size_t cnt = 0;
      for (size_t j = 0; j < static_cast<size_t>(img.height); j++) {
         for (size_t i = 0; i < static_cast<size_t>(img.width); i++) {
            float y = 2.0 * ((type_t)j / (type_t)img.height) - 1.0;
            float x = 2.0 * ((type_t)i / (type_t)img.width) - 1.0;
            pos_temp.set_value(cnt, 0, y);
            pos_temp.set_value(cnt, 1, x);
            val.set_value(cnt, 0, reinterpret_cast<pixel*>(img.data)[i + j * img.width] / 255.0);
            cnt++;
         }
      }

      NumericMatrix::HostMatrix<type_t> ff_input = generate_fourier_features<type_t>(pos_temp, 32, 10.0);

      NumericMatrix::Matrix<type_t, HW> pos(n_samples, ff_input.ncols(), &p);

      NumericMatrix::get_from_host(pos, ff_input);

      NumericMatrix::Matrix<type_t, HW> pos2 = pos;
      NumericMatrix::Matrix<type_t, HW> val2 = val;
      std::vector<int> arch{200, 200, 1};

      NeuralNetwork<type_t, HW, ACTIVATION::RELU> nn(arch, &p, pos, val, BATCHSIZE);
      auto V = std::chrono::high_resolution_clock::now();
      for (size_t i = 0; i < 5; i++) {
         auto l = nn.train(BATCHSIZE, 1e-3);
      }

      auto Y = std::chrono::high_resolution_clock::now();
      auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(Y - V).count();
      hipDeviceSynchronize();

      nn.evaluate(pos2, recon);
      cnt = 0;
      for (size_t j = 0; j < static_cast<size_t>(img.height); j++) {
         for (size_t i = 0; i < static_cast<size_t>(img.width); i++) {
            auto val = std::clamp(recon.get_value(cnt, 0), 0.0f, 1.0f);
            reinterpret_cast<pixel*>(rec_img.data)[i + j * img.width] = val * 255.0;
            cnt++;
         }
      }
      double psnr =
          calculate_psnr((unsigned char*)img.data, (unsigned char*)rec_img.data, rec_img.width, rec_img.height);
      EXPECT_TRUE(psnr>20.0 );

   }
   p.defrag();

// CleanUp
#ifdef USE_GPU
   hipFree(mem);
#else
   free(mem);
#endif
   return 0;
}
